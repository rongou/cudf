// build:
// nvcc -lcuda -lcufile cufile_bug.cu
//
// run:
// ./a.out


#include <hip/hip_runtime.h>
#include <cassert>

#include <cufile.h>
#include <fcntl.h>
#include <unistd.h>

#include <sys/stat.h>

#define DEVICE 0
#define SIZE 4097
#define OUT "/tmp/cufile"

int main() {
  assert(hipSetDevice(DEVICE) == hipSuccess);
  assert(hipFree(0) == hipSuccess);

  void *buffer;
  assert(hipMalloc(&buffer, SIZE) == hipSuccess);
  assert(hipMemset(buffer, 42, SIZE) == hipSuccess);

  assert(cuFileDriverOpen().err == CU_FILE_SUCCESS);

  auto const file_descriptor = open(OUT, O_CREAT | O_WRONLY | O_DIRECT, S_IRUSR | S_IWUSR);
  assert(file_descriptor >= 0);
  CUfileDescr_t cufile_descriptor{CU_FILE_HANDLE_TYPE_OPAQUE_FD, file_descriptor};
  CUfileHandle_t cufile_handle{};
  assert(cuFileHandleRegister(&cufile_handle, &cufile_descriptor).err == CU_FILE_SUCCESS);

  hipCtx_t ctx0;
  assert(hipCtxGetCurrent(&ctx0) == hipSuccess);

  assert(cuFileWrite(cufile_handle, buffer, SIZE, 0, 0) == SIZE);

  hipCtx_t ctx1;
  assert(hipCtxGetCurrent(&ctx1) == hipSuccess);
  assert(ctx0 == ctx1);

  assert(hipFree(buffer) == hipSuccess);
  cuFileHandleDeregister(cufile_handle);
  assert(close(file_descriptor) == 0);

  assert(cuFileDriverClose().err == CU_FILE_SUCCESS);

  return 0;
}
