// build:
// nvcc --default-stream per-thread -lcufile cufile_bug.cu
//
// run:
// ./a.out


#include <hip/hip_runtime.h>
#include <cassert>
#include <string>
#include <thread>

#include <cufile.h>
#include <fcntl.h>
#include <unistd.h>

#include <sys/stat.h>

#define DEVICE 0
#define SIZE 1 << 30
#define BASE "/tmp/cufile"

void write_buffers(void **buffers) {
  for (int i = 0; i < 2; i++) {
    assert(hipSetDevice(DEVICE) == hipSuccess);
    auto const file_descriptor = open((BASE + std::to_string(i)).c_str(),
                                      O_CREAT | O_WRONLY | O_DIRECT, S_IRUSR | S_IWUSR);
    assert(file_descriptor >= 0);
    CUfileDescr_t cufile_descriptor{CU_FILE_HANDLE_TYPE_OPAQUE_FD, file_descriptor};
    CUfileHandle_t cufile_handle{};
    assert(cuFileHandleRegister(&cufile_handle, &cufile_descriptor).err == CU_FILE_SUCCESS);

    assert(cuFileWrite(cufile_handle, buffers[i], SIZE, 0, 0) == SIZE);

    assert(hipFree(buffers[i]) == hipSuccess);
    cuFileHandleDeregister(cufile_handle);
    assert(close(file_descriptor) == 0);
  }
}

int main() {
  assert(hipSetDevice(DEVICE) == hipSuccess);
  assert(hipFree(0) == hipSuccess);

  void *buffers[2];
  for (int i = 0; i < 2; i++) {
    assert(hipMalloc(&buffers[i], SIZE) == hipSuccess);
    assert(hipMemset(buffers[i], i, SIZE) == hipSuccess);
  }

  assert(cuFileDriverOpen().err == CU_FILE_SUCCESS);
  std::thread t{write_buffers, buffers};
  t.join();
  assert(cuFileDriverClose().err == CU_FILE_SUCCESS);

  return 0;
}
